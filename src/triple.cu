#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <stdbool.h>
#include <time.h>
//#include <omp.h>

#include <cstdlib>  // for malloc
#include <cstdio>  // for io



// cuda_kernels.cuh
#ifndef CUDA_KERNELS_H
#define CUDA_KERNELS_H

#include <hip/hip_runtime.h>

__global__ void calculateSwapDeltasKernel(
    const short* adj_matrix_m,
    const short* adj_matrix_f,
    const int* mapping,
    int num_nodes,
    int* deltas,
    int* node_pairs
);

#endif

// cuda_kernels.cu
//#include "cuda_kernels.cuh"



// Add these constants at the top
#define BATCH_SIZE  270000  // Adjust based on GPU memory
#define NUM_PERMUTATIONS 6  // 3! possible permutations


__global__ void calculateThreeNodeSwapKernel(
    const short* adj_matrix_m,
    const short* adj_matrix_f,
    const int* mapping,
    int num_nodes,
    long int batch_start,
    int batch_size,
    int* deltas,
    int* node_triplets
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size) return;
    
    // Convert global index to triplet index
    long int global_idx = batch_start + idx;
    int n = num_nodes;
    int matrix_dim = num_nodes + 1;
    
    // Convert linear index to three nodes
    int node_m1 = 1;
    long int temp1 = (long int) (n-1) * (n-2) / 6;
    while (global_idx >= temp1) {
        global_idx -= temp1;
        node_m1++;
        n--;
        temp1 = (long int) (n-1) * (n-2) / 6;
    }
    
    int node_m2 = node_m1 + 1;
    temp1 = n - 2;
    while (global_idx >= temp1) {
        global_idx -= temp1;
        node_m2++;
        temp1--;
    }
    
    int node_m3 = node_m2 + global_idx + 1;
    
    // Get current mappings
    int node_f1 = mapping[node_m1];
    int node_f2 = mapping[node_m2];
    int node_f3 = mapping[node_m3];
    
    // Try all 6 possible permutations and find the best one
    int permutations[6][3] = {
        {node_f1, node_f2, node_f3},  // Original
        {node_f1, node_f3, node_f2},  // Swap 2,3
        {node_f2, node_f1, node_f3},  // Swap 1,2
        {node_f2, node_f3, node_f1},  // 231
        {node_f3, node_f1, node_f2},  // 312
        {node_f3, node_f2, node_f1}   // 321
    };
    
    int best_delta = 0;
    int best_perm = 0;
    
    // Calculate delta for each permutation
    for (int p = 1; p < 6; p++) {  // Start from 1 as 0 is original mapping
        int delta = 0;
        int new_f1 = permutations[p][0];
        int new_f2 = permutations[p][1];
        int new_f3 = permutations[p][2];
        
        // Calculate delta for node_m1 connections to rest of graph
        for (int i = 1; i <= num_nodes; i++) {
            if (i == node_m1 || i == node_m2 || i == node_m3) continue;
            
            // Outgoing edges from node_m1
            if (adj_matrix_m[node_m1 * matrix_dim + i] > 0) {
                int weight = adj_matrix_m[node_m1 * matrix_dim + i];
                int dst_f = mapping[i];
                delta += min(weight, adj_matrix_f[new_f1 * matrix_dim + dst_f]) -
                        min(weight, adj_matrix_f[node_f1 * matrix_dim + dst_f]);
            }
            
            // Incoming edges to node_m1
            if (adj_matrix_m[i * matrix_dim + node_m1] > 0) {
                int weight = adj_matrix_m[i * matrix_dim + node_m1];
                int src_f = mapping[i];
                delta += min(weight, adj_matrix_f[src_f * matrix_dim + new_f1]) -
                        min(weight, adj_matrix_f[src_f * matrix_dim + node_f1]);
            }
        }
        
        // Calculate delta for node_m2 connections to rest of graph
        for (int i = 1; i <= num_nodes; i++) {
            if (i == node_m1 || i == node_m2 || i == node_m3) continue;
            
            // Outgoing edges from node_m2
            if (adj_matrix_m[node_m2 * matrix_dim + i] > 0) {
                int weight = adj_matrix_m[node_m2 * matrix_dim + i];
                int dst_f = mapping[i];
                delta += min(weight, adj_matrix_f[new_f2 * matrix_dim + dst_f]) -
                        min(weight, adj_matrix_f[node_f2 * matrix_dim + dst_f]);
            }
            
            // Incoming edges to node_m2
            if (adj_matrix_m[i * matrix_dim + node_m2] > 0) {
                int weight = adj_matrix_m[i * matrix_dim + node_m2];
                int src_f = mapping[i];
                delta += min(weight, adj_matrix_f[src_f * matrix_dim + new_f2]) -
                        min(weight, adj_matrix_f[src_f * matrix_dim + node_f2]);
            }
        }
        
        // Calculate delta for node_m3 connections to rest of graph
        for (int i = 1; i <= num_nodes; i++) {
            if (i == node_m1 || i == node_m2 || i == node_m3) continue;
            
            // Outgoing edges from node_m3
            if (adj_matrix_m[node_m3 * matrix_dim + i] > 0) {
                int weight = adj_matrix_m[node_m3 * matrix_dim + i];
                int dst_f = mapping[i];
                delta += min(weight, adj_matrix_f[new_f3 * matrix_dim + dst_f]) -
                        min(weight, adj_matrix_f[node_f3 * matrix_dim + dst_f]);
            }
            
            // Incoming edges to node_m3
            if (adj_matrix_m[i * matrix_dim + node_m3] > 0) {
                int weight = adj_matrix_m[i * matrix_dim + node_m3];
                int src_f = mapping[i];
                delta += min(weight, adj_matrix_f[src_f * matrix_dim + new_f3]) -
                        min(weight, adj_matrix_f[src_f * matrix_dim + node_f3]);
            }
        }
        
        // Calculate deltas between the three nodes themselves
        // m1 -> m2
        int w12 = adj_matrix_m[node_m1 * matrix_dim + node_m2];
        if (w12 > 0) {
            delta += min(w12, adj_matrix_f[new_f1 * matrix_dim + new_f2]) -
                    min(w12, adj_matrix_f[node_f1 * matrix_dim + node_f2]);
        }
        
        // m2 -> m1
        int w21 = adj_matrix_m[node_m2 * matrix_dim + node_m1];
        if (w21 > 0) {
            delta += min(w21, adj_matrix_f[new_f2 * matrix_dim + new_f1]) -
                    min(w21, adj_matrix_f[node_f2 * matrix_dim + node_f1]);
        }
        
        // m2 -> m3
        int w23 = adj_matrix_m[node_m2 * matrix_dim + node_m3];
        if (w23 > 0) {
            delta += min(w23, adj_matrix_f[new_f2 * matrix_dim + new_f3]) -
                    min(w23, adj_matrix_f[node_f2 * matrix_dim + node_f3]);
        }
        
        // m3 -> m2
        int w32 = adj_matrix_m[node_m3 * matrix_dim + node_m2];
        if (w32 > 0) {
            delta += min(w32, adj_matrix_f[new_f3 * matrix_dim + new_f2]) -
                    min(w32, adj_matrix_f[node_f3 * matrix_dim + node_f2]);
        }
        
        // m3 -> m1
        int w31 = adj_matrix_m[node_m3 * matrix_dim + node_m1];
        if (w31 > 0) {
            delta += min(w31, adj_matrix_f[new_f3 * matrix_dim + new_f1]) -
                    min(w31, adj_matrix_f[node_f3 * matrix_dim + node_f1]);
        }
        
        // m1 -> m3
        int w13 = adj_matrix_m[node_m1 * matrix_dim + node_m3];
        if (w13 > 0) {
            delta += min(w13, adj_matrix_f[new_f1 * matrix_dim + new_f3]) -
                    min(w13, adj_matrix_f[node_f1 * matrix_dim + node_f3]);
        }
        
        if (delta > best_delta) {
            best_delta = delta;
            best_perm = p;
        }
    }
    
    deltas[idx] = best_delta;
    node_triplets[idx * 4] = node_m1;
    node_triplets[idx * 4 + 1] = node_m2;
    node_triplets[idx * 4 + 2] = node_m3;
    node_triplets[idx * 4 + 3] = best_perm;
}



__global__ void calculateSwapDeltasKernel(
    const short* adj_matrix_m,
    const short* adj_matrix_f,
    const int* mapping,
    int num_nodes,
    int* deltas,
    int* node_pairs
) {




    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int num_pairs = (num_nodes * (num_nodes - 1)) / 2;

    if (idx >= num_pairs) return;

    // Convert linear index to vertex pair
    int node_m1 = 1;
    int temp = num_nodes - 1;
    while (idx >= temp) {
        idx -= temp;
        node_m1++;
        temp--;
    }
    int node_m2 = node_m1 + idx + 1;

    int matrix_dim = num_nodes + 1;  // Add this line
    int delta = 0;
    int node_f1 = mapping[node_m1];
    int node_f2 = mapping[node_m2];



    // Update all matrix indexing to use matrix_dim
    for (int i = 1; i <= num_nodes; i++) {
        if (i == node_m1 || i == node_m2) continue;

        // Outgoing edges from node_m1
        if (adj_matrix_m[node_m1 * matrix_dim + i] > 0) {
            int weight_m = adj_matrix_m[node_m1 * matrix_dim + i];
            int dst_f = mapping[i];
            delta += min(weight_m, adj_matrix_f[node_f2 * matrix_dim + dst_f]) -
                    min(weight_m, adj_matrix_f[node_f1 * matrix_dim + dst_f]);




        }
        // Update other matrix accesses similarly...


        // Incoming edges to node_m1
        if (adj_matrix_m[i * matrix_dim + node_m1] > 0) {
            int weight_m = adj_matrix_m[i * matrix_dim + node_m1];
            int src_f = mapping[i];
            delta += min(weight_m, adj_matrix_f[src_f * matrix_dim + node_f2]) - 
                    min(weight_m, adj_matrix_f[src_f * matrix_dim + node_f1]);
        }
        
        // Outgoing edges from node_m2
        if (adj_matrix_m[node_m2 * matrix_dim + i] > 0) {
            int weight_m = adj_matrix_m[node_m2 * matrix_dim + i];
            int dst_f = mapping[i];
            delta += min(weight_m, adj_matrix_f[node_f1 * matrix_dim + dst_f]) - 
                    min(weight_m, adj_matrix_f[node_f2 * matrix_dim + dst_f]);
        }
        
        // Incoming edges to node_m2
        if (adj_matrix_m[i * matrix_dim + node_m2] > 0) {
            int weight_m = adj_matrix_m[i * matrix_dim + node_m2];
            int src_f = mapping[i];
            delta += min(weight_m, adj_matrix_f[src_f * matrix_dim + node_f1]) - 
                    min(weight_m, adj_matrix_f[src_f * matrix_dim + node_f2]);
        }
    }
    
    // Handle direct edges between swapped nodes
    int m1_to_m2 = adj_matrix_m[node_m1 * matrix_dim + node_m2];
    if (m1_to_m2 > 0) {
        delta += min(m1_to_m2, adj_matrix_f[node_f2 * matrix_dim + node_f1]) - 
                min(m1_to_m2, adj_matrix_f[node_f1 * matrix_dim + node_f2]);
    }
    
    int m2_to_m1 = adj_matrix_m[node_m2 * matrix_dim + node_m1];
    if (m2_to_m1 > 0) {
        delta += min(m2_to_m1, adj_matrix_f[node_f1 * matrix_dim + node_f2]) - 
                min(m2_to_m1, adj_matrix_f[node_f2 * matrix_dim + node_f1]);
    }
    
    deltas[idx] = delta;
    node_pairs[idx * 2] = node_m1;
    node_pairs[idx * 2 + 1] = node_m2;
}




// Add these constants
#define SYNC_INTERVAL 1800  // Sync every 1 hour
#define TAG_SCORE 1
#define TAG_MAPPING 2
#define TAG_TERMINATE 3

#define SAVE_INTERVAL 4200
#define UPDATE_INTERVAL 4000
#define MAX_LINE_LENGTH 1024
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))
#define MAX_NODES 100000

const int NUM_NODES = 18524;

typedef enum {
    LOG_LEVEL_INFO = 0,
    LOG_LEVEL_DEBUG = 1,
    LOG_LEVEL_ERROR = 2
} LogLevel;

#define CURRENT_LOG_LEVEL LOG_LEVEL_INFO

#define LOG_ERROR(fmt, ...) \
    if (CURRENT_LOG_LEVEL <= LOG_LEVEL_ERROR) { \
        fprintf(stderr, "[ERROR] %s:%d: " fmt "\n", __func__, __LINE__, ##__VA_ARGS__); \
    }

#define LOG_INFO(fmt, ...) \
    if (CURRENT_LOG_LEVEL <= LOG_LEVEL_INFO) { \
        printf("[INFO] " fmt "\n", ##__VA_ARGS__); \
    }

#define LOG_DEBUG(fmt, ...) \
    if (CURRENT_LOG_LEVEL <= LOG_LEVEL_DEBUG) { \
        printf("[DEBUG] %s: " fmt "\n", __func__, ##__VA_ARGS__); \
    }







// Structure definitions
typedef struct EdgeMap {
    int* to_nodes;
    int* weights;
    int count;
    int capacity;
} EdgeMap;

typedef struct Graph {
    EdgeMap* edges;
    EdgeMap* reverse_edges;
    short** adj_matrix;
    int node_capacity;
} Graph;

typedef struct NodeMetrics {
    int in_degree;
    int out_degree;
    int total_weight;
    double avg_in_weight;
    double avg_out_weight;
    int ordering_rank;
} NodeMetrics;


// Progress bar function
void print_progress(int current, int total, const char* prefix) {
    const int bar_width = 50;
    float progress = (float)current / total;
    int filled = (int)(bar_width * progress);
    
    printf("\r%s [", prefix);
    for (int i = 0; i < bar_width; i++) {
        if (i < filled) printf("=");
        else if (i == filled) printf(">");
        else printf(" ");
    }
    printf("] %.1f%%", progress * 100);
    fflush(stdout);
    if (current == total) printf("\n");
}

// Function to format numbers with commas
char* format_number(int num) {
    static char formatted[32];
    char temp[32];
    int i = 0, j = 0;
    
    sprintf(temp, "%d", num);
    int len = strlen(temp);
    
    while (len > 0) {
        if (i > 0 && i % 3 == 0) formatted[j++] = ',';
        formatted[j++] = temp[len - 1];
        len--;
        i++;
    }
    formatted[j] = '\0';
    
    // Reverse the string
    for (i = 0; i < j/2; i++) {
        char t = formatted[i];
        formatted[i] = formatted[j-1-i];
        formatted[j-1-i] = t;
    }
    
    return formatted;
}

EdgeMap* new_edge_map() {
    EdgeMap* em = (EdgeMap*)malloc(sizeof(EdgeMap));
    if (!em) {
        LOG_ERROR("Failed to allocate EdgeMap");
        exit(1);
    }
    em->capacity = 100;
    em->count = 0;
    em->to_nodes =(int *) malloc(sizeof(int) * em->capacity);
    em->weights = (int *) malloc(sizeof(int) * em->capacity);
    if (!em->to_nodes || !em->weights) {
        LOG_ERROR("Failed to allocate EdgeMap arrays");
        exit(1);
    }
    return em;
}

Graph* new_graph() {
    Graph* g =(Graph *) malloc(sizeof(Graph));
    if (!g) {
        LOG_ERROR("Failed to allocate Graph");
        exit(1);
    }
    g->edges = NULL;
    g->reverse_edges = NULL;
    g->adj_matrix = NULL;
    g->adj_matrix = (short**)malloc((NUM_NODES+1) * sizeof(short*));
    for (int i = 0; i <= NUM_NODES; ++i) {
        g->adj_matrix[i] = (short*)calloc((NUM_NODES+1), sizeof(short)); // Initialize to 0
    }

    //g->nodes = malloc(sizeof(int) * 10000);
    //if (!g->nodes) {
    //    LOG_ERROR("Failed to allocate nodes array");
    //    exit(1);
    //}
    //g->node_count = 0;
    g->node_capacity = 10000;
    return g;
}

void add_to_edge_map(EdgeMap* em, int to, int weight) {
    if (em->count >= em->capacity) {
        em->capacity *= 2;
        int* new_to_nodes = (int *) realloc(em->to_nodes, sizeof(int) * em->capacity);
        int* new_weights = (int *) realloc(em->weights, sizeof(int) * em->capacity);
        if (!new_to_nodes || !new_weights) {
            LOG_ERROR("Failed to reallocate EdgeMap arrays");
            exit(1);
        }
        em->to_nodes = new_to_nodes;
        em->weights = new_weights;
    }
    em->to_nodes[em->count] = to;
    em->weights[em->count] = weight;
    em->count++;
}

//void add_node(Graph* g, int node) {
//    for (int i = 0; i < g->node_count; i++) {
//        if (g->nodes[i] == node) return;
//    }
//    
//    if (g->node_count >= g->node_capacity) {
//        g->node_capacity *= 2;
//        int* new_nodes = realloc(g->nodes, sizeof(int) * g->node_capacity);
//        if (!new_nodes) {
//            LOG_ERROR("Failed to reallocate nodes array");
//            exit(1);
//        }
//        g->nodes = new_nodes;
//    }
//    g->nodes[g->node_count++] = node;
//}

void add_edge(Graph* g, int from, int to, int weight) {
  //    LOG_DEBUG("Adding edge: %d -> %d (weight: %d)", from, to, weight);
    g->adj_matrix[from][to] = weight;
    if (g->edges == NULL) {
        g->edges = (EdgeMap*) calloc(MAX_NODES, sizeof(EdgeMap));
        g->reverse_edges = (EdgeMap*)calloc(MAX_NODES, sizeof(EdgeMap));
        if (!g->edges || !g->reverse_edges) {
            LOG_ERROR("Failed to allocate edges arrays");
            exit(1);
        }
    }
    
    if (g->edges[from].count == 0) {
        g->edges[from] = *new_edge_map();
    }
    if (g->reverse_edges[to].count == 0) {
        g->reverse_edges[to] = *new_edge_map();
    }
    
    add_to_edge_map(&g->edges[from], to, weight);
    add_to_edge_map(&g->reverse_edges[to], from, weight);
}

int get_weight(Graph* g, int from, int to) {
    return g->adj_matrix[from][to];
    //for (int i = 0; i < g->edges[from].count; i++) {
    //    if (g->edges[from].to_nodes[i] == to) {
    //        return g->edges[from].weights[i];
    //    }
    //}
    //return 0;
}

int* read_ordering(const char* filename, int max_node) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        LOG_ERROR("Error opening file: %s", filename);
        exit(1);
    }

    int* ordering = (int *) calloc(max_node + 1, sizeof(int));
    char line[MAX_LINE_LENGTH];
    
    LOG_INFO("Reading ordering from %s", filename);
    
    // Skip header
    fgets(line, MAX_LINE_LENGTH, file);
    
    while (fgets(line, MAX_LINE_LENGTH, file)) {
        int node_id, order;
        sscanf(line, "%d,%d", &node_id, &order);
        ordering[node_id] = order;
    }
    
    fclose(file);
    return ordering;
}

NodeMetrics* calculate_node_metrics(Graph* g, const char* ordering_path) {
    int max_node = NUM_NODES;
    //for (int i = 0; i < g->node_count; i++) {
    //    if (g->nodes[i] > max_node) max_node = g->nodes[i];
    //}

    LOG_INFO("Calculating node metrics");
    
    int* ordering = (int *) read_ordering(ordering_path, max_node);
    NodeMetrics* metrics = ( NodeMetrics*) calloc(max_node + 1, sizeof(NodeMetrics));
    
    for (int node=1; node<=NUM_NODES; node++) {
        //int node = g->nodes[i];
        NodeMetrics* m = &metrics[node];
        
        m->out_degree = g->edges[node].count;
        for (int j = 0; j < g->edges[node].count; j++) {
            m->total_weight += g->edges[node].weights[j];
        }
        
        m->in_degree = g->reverse_edges[node].count;
        for (int j = 0; j < g->reverse_edges[node].count; j++) {
            m->total_weight += g->reverse_edges[node].weights[j];
        }
        
        if (m->out_degree > 0) {
            m->avg_out_weight = (double)m->total_weight / m->out_degree;
        }
        if (m->in_degree > 0) {
            m->avg_in_weight = (double)m->total_weight / m->in_degree;
        }
        
        m->ordering_rank = ordering[node];
    }
    
    free(ordering);
    return metrics;
}

double calculate_node_similarity(NodeMetrics m1, NodeMetrics m2) {
    double score = 5 * fabs(m1.in_degree - m2.in_degree) +
                   5 * fabs(m1.out_degree - m2.out_degree) +
                   fabs(m1.avg_in_weight - m2.avg_in_weight) +
                   fabs(m1.avg_out_weight - m2.avg_out_weight);
    
    double ordering_sim = fabs(m1.ordering_rank - m2.ordering_rank);
    score = 0.7 * score + 0.3 * ordering_sim;
    
    return -score;
}

int calculate_alignment_score(Graph* gm, Graph* gf, int* mapping) {
    int score = 0;
    
    for (int src_m = 1; src_m <= NUM_NODES; src_m++) {
        //int src_m = gm->i;
        for (int j = 0; j < gm->edges[src_m].count; j++) {
            int dst_m = gm->edges[src_m].to_nodes[j];
            int weight_m = gm->edges[src_m].weights[j];
            int src_f = mapping[src_m];
            int dst_f = mapping[dst_m];
            score += MIN(weight_m, gf->adj_matrix[src_f][dst_f]);
        }
    }
    
    return score;
}

void validate_mapping_changes(int* old_mapping, int* new_mapping, int max_node,
                            int node_m1, int node_m2) {
    for (int i = 1; i <= max_node; i++) {
        if (i != node_m1 && i != node_m2) {
            if (old_mapping[i] != new_mapping[i]) {
                LOG_ERROR("Unexpected mapping change for node %d: %d -> %d",
                         i, old_mapping[i], new_mapping[i]);
            }
        }
    }
}

int calculate_swap_delta(Graph* gm, Graph* gf, int* mapping, int node_m1, int node_m2) {
    int node_f1 = mapping[node_m1];
    int node_f2 = mapping[node_m2];
    int delta = 0;
    
    // Handle outgoing edges from node_m1
    for (int i = 0; i < gm->edges[node_m1].count; i++) {
        int dst_m = gm->edges[node_m1].to_nodes[i];
        if (dst_m == node_m2) continue;  // Skip direct edge between swapped nodes
        
        int weight_m = gm->edges[node_m1].weights[i];
        int dst_f = mapping[dst_m];
        
        // Remove old contribution
        int old_weight = MIN(weight_m, gf->adj_matrix[node_f1][dst_f]);
        // Add new contribution
        int new_weight = MIN(weight_m, gf->adj_matrix[node_f2][dst_f]);
        
        delta += new_weight - old_weight;
    }
    
    // Handle incoming edges to node_m1
    for (int i = 0; i < gm->reverse_edges[node_m1].count; i++) {
        int src_m = gm->reverse_edges[node_m1].to_nodes[i];
        if (src_m == node_m2) continue;  // Skip direct edge between swapped nodes
        
        int weight_m = gm->reverse_edges[node_m1].weights[i];
        int src_f = mapping[src_m];
        
        // Remove old contribution
        int old_weight = MIN(weight_m, gf->adj_matrix[src_f][node_f1]);
        // Add new contribution
        int new_weight = MIN(weight_m, gf->adj_matrix[src_f][node_f2]);
        
        delta += new_weight - old_weight;
    }
    
    // Handle outgoing edges from node_m2
    for (int i = 0; i < gm->edges[node_m2].count; i++) {
        int dst_m = gm->edges[node_m2].to_nodes[i];
        if (dst_m == node_m1) continue;  // Skip direct edge between swapped nodes
        
        int weight_m = gm->edges[node_m2].weights[i];
        int dst_f = mapping[dst_m];
        
        // Remove old contribution
        int old_weight = MIN(weight_m, gf->adj_matrix[node_f2][dst_f]);
        // Add new contribution
        int new_weight = MIN(weight_m, gf->adj_matrix[node_f1][dst_f]);
        
        delta += new_weight - old_weight;
    }
    
    // Handle incoming edges to node_m2
    for (int i = 0; i < gm->reverse_edges[node_m2].count; i++) {
        int src_m = gm->reverse_edges[node_m2].to_nodes[i];
        if (src_m == node_m1) continue;  // Skip direct edge between swapped nodes
        
        int weight_m = gm->reverse_edges[node_m2].weights[i];
        int src_f = mapping[src_m];
        
        // Remove old contribution
        int old_weight = MIN(weight_m, gf->adj_matrix[src_f][node_f2]);
        // Add new contribution
        int new_weight = MIN(weight_m, gf->adj_matrix[src_f][node_f1]);
        
        delta += new_weight - old_weight;
    }
    
    // Handle direct edges between the swapped nodes
    // From m1 to m2
    int m1_to_m2 = gm->adj_matrix[node_m1][node_m2];
    if (m1_to_m2 > 0) {
        int old_weight = MIN(m1_to_m2, gf->adj_matrix[node_f1][node_f2]);
        int new_weight = MIN(m1_to_m2, gf->adj_matrix[node_f2][node_f1]);
        delta += new_weight - old_weight;
    }
    
    // From m2 to m1
    int m2_to_m1 = gm->adj_matrix[node_m2][node_m1];
    if (m2_to_m1 > 0) {
        int old_weight = MIN(m2_to_m1, gf->adj_matrix[node_f2][node_f1]);
        int new_weight = MIN(m2_to_m1, gf->adj_matrix[node_f1][node_f2]);
        delta += new_weight - old_weight;
    }
    
    return delta;
}

void write_mapping(const char* filename, int* mapping, int max_node) {
    FILE* file = fopen(filename, "w");
    if (!file) {
        LOG_ERROR("Error creating file: %s", filename);
        exit(1);
    }
    
    fprintf(file, "Male Node ID,Female Node ID\n");
    for (int i = 1; i <= max_node; i++) {
        if (mapping[i] != 0) {
            fprintf(file, "m%d,f%d\n", i, mapping[i]);
        }
    }
    
    fclose(file);
}

// Function to load benchmark mapping from CSV
int* load_benchmark_mapping(const char* filename, int max_node) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        LOG_ERROR("Failed to open file: %s", filename);
        return NULL;
    }
    
    int* mapping = (int *) calloc(max_node + 1, sizeof(int));
    char line[MAX_LINE_LENGTH];
    int count = 0;
    
    // Try to get expected score from filename
    int expected_score = 0;
    const char* underscore = strrchr(filename, '_');
    if (underscore) {
        expected_score = atoi(underscore + 1);
        LOG_INFO("Expected score from filename: %d", expected_score);
    }
    
    // Skip header
    fgets(line, MAX_LINE_LENGTH, file);
    
    LOG_INFO("Loading benchmark mapping from %s", filename);
    
    while (fgets(line, MAX_LINE_LENGTH, file)) {
        int male_id, female_id;
        
        // Try direct integer format first
        if (sscanf(line, "%d,%d", &male_id, &female_id) == 2) {
            mapping[male_id] = female_id;
            count++;
        } else {
            // Try format with prefixes
            char male_str[20], female_str[20];
            if (sscanf(line, "%[^,],%s", male_str, female_str) == 2) {
                male_id = atoi(male_str + (male_str[0] == 'm' ? 1 : 0));
                female_id = atoi(female_str + (female_str[0] == 'f' ? 1 : 0));
                mapping[male_id] = female_id;
                count++;
            }
        }
    }
    
    //LOG_INFO("Loaded %s mappings", format_number(count));
    
    fclose(file);
    return mapping;
}

// Save intermediate mapping with verification
void save_intermediate_mapping(const char* filename, int* mapping, int max_node, 
                             Graph* gm, Graph* gf, int current_score) {
    write_mapping(filename, mapping, max_node);
    
    // Verify written mapping
    int* verification = load_benchmark_mapping(filename, max_node);
    if (verification) {
        int verify_score = calculate_alignment_score(gm, gf, verification);
        if (verify_score != current_score) {
            LOG_ERROR("Score mismatch - internal: %d, written: %d", 
                     current_score, verify_score);
        }
        free(verification);
    }
}

void random_swap_k_vertices(int* mapping, int n, int k,int seed) {
    // Create array for tracking selected vertices
    int* selected = (int*)malloc(2*k * sizeof(int));
    int* used = (int*)calloc(n, sizeof(int));  // Track used positions

    if (k*2>n) {
            LOG_ERROR("K is too big : %d",k); 
    } 
    // Randomly select k vertices
    int count = 0;
    srand(time(NULL)+seed);
    while (count < k*2) {
        int idx = rand() % n;
        if (!used[idx]) {
            selected[count] = idx;
            used[idx] = 1;
            count++;
        }
    }

    // Store original values
    int* original_values = (int*)malloc(2*k * sizeof(int));
    for (int i = 0; i <2* k; i++) {
        original_values[i] = mapping[selected[i]];

    }

    for (int i = 0; i < k; i++) {
        mapping[selected[i]] = original_values[i+k];
        mapping[selected[i+k]] = original_values[i];
    }

    // Clean up
    free(selected);
    free(used);
    free(original_values);
}




// Add this function
void verify_matrix_copy(Graph* g, int max_node) {
    short* h_adj_matrix = (short*)malloc((max_node + 1) * (max_node + 1) * sizeof(short));

    // Count non-zero entries in original
    int orig_nonzero = 0;
    for (int i = 1; i <= max_node; i++) {
        for (int j = 1; j <= max_node; j++) {
            if (g->adj_matrix[i][j] > 0) {
                orig_nonzero++;
                printf("Found edge %d->%d weight=%d\n",
                       i, j, g->adj_matrix[i][j]);
            }
            h_adj_matrix[i * (max_node + 1) + j] = g->adj_matrix[i][j];
        }
    }

    // Verify copied matrix
    int copy_nonzero = 0;
    for (int i = 1; i <= max_node; i++) {
        for (int j = 1; j <= max_node; j++) {
            if (h_adj_matrix[i * (max_node + 1) + j] > 0) {
                copy_nonzero++;
            }
        }
    }

    printf("Original matrix had %d edges, copied matrix has %d edges\n",
           orig_nonzero, copy_nonzero);

    free(h_adj_matrix);
}







// Add to optimize_mapping function
void apply_permutation(int* mapping, int node1, int node2, int node3, int perm_idx) {
    int f1 = mapping[node1];
    int f2 = mapping[node2];
    int f3 = mapping[node3];
    
    // Permutation mappings based on perm_idx
    switch(perm_idx) {
        case 1: // f1,f3,f2
            mapping[node2] = f3;
            mapping[node3] = f2;
            break;
        case 2: // f2,f1,f3
            mapping[node1] = f2;
            mapping[node2] = f1;
            break;
        case 3: // f2,f3,f1
            mapping[node1] = f2;
            mapping[node2] = f3;
            mapping[node3] = f1;
            break;
        case 4: // f3,f1,f2
            mapping[node1] = f3;
            mapping[node2] = f1;
            mapping[node3] = f2;
            break;
        case 5: // f3,f2,f1
            mapping[node1] = f3;
            mapping[node2] = f2;
            mapping[node3] = f1;
            break;
    }
}





// Modified optimize_mapping function
int* optimize_mapping(Graph* gm, Graph* gf, int* initial_mapping, const char* out_path) {




    // Check if CUDA is already initialized
    int device = -1;
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess) {
        LOG_ERROR("CUDA not initialized: %s", hipGetErrorString(err));
        return NULL;
    }
    printf("Using CUDA device %d\n", device);
    // Print device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using CUDA device: %s\n", prop.name);






    int rank=0;
    int size=4;
    
    int max_node = NUM_NODES;
    
    // Allocate and initialize mappings
    int* current_mapping = (int*)malloc((max_node + 1) * sizeof(int));
    int* best_mapping = (int*)malloc((max_node + 1) * sizeof(int));
    int* old_mapping = (int*)malloc((max_node + 1) * sizeof(int));
    memcpy(current_mapping, initial_mapping, (max_node + 1) * sizeof(int));
    memcpy(best_mapping, current_mapping, (max_node + 1) * sizeof(int));
    
    // Prepare GPU data structures
    short *d_adj_matrix_m, *d_adj_matrix_f;
    int *d_mapping, *d_deltas, *d_node_pairs;
    



    // Allocate GPU memory
    hipMalloc(&d_adj_matrix_m, (max_node + 1) * (max_node + 1) * sizeof(short));
    hipMalloc(&d_adj_matrix_f, (max_node + 1) * (max_node + 1) * sizeof(short));
    hipMalloc(&d_mapping, (max_node + 1) * sizeof(int));
    
    // Convert adjacency matrices to linear arrays and copy to GPU
    short* h_adj_matrix_m = (short*)malloc((max_node + 1) * (max_node + 1) * sizeof(short));
    short* h_adj_matrix_f = (short*)malloc((max_node + 1) * (max_node + 1) * sizeof(short));
    
    for (int i = 1; i <= max_node; i++) {
        for (int j = 1; j <= max_node; j++) {
            h_adj_matrix_m[i * (max_node + 1) + j] = gm->adj_matrix[i][j];
            h_adj_matrix_f[i * (max_node + 1) + j] = gf->adj_matrix[i][j];
        }
    }
    




    hipMemcpy(d_adj_matrix_m, h_adj_matrix_m, (max_node + 1) * (max_node + 1) * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(d_adj_matrix_f, h_adj_matrix_f, (max_node + 1) * (max_node + 1) * sizeof(short), hipMemcpyHostToDevice);
    


    // In the main optimization loop:
    long int total_triplets = ((long int) max_node * (max_node - 1) * (max_node - 2)) / 6;
   
    printf("total triplets =%ld\n",total_triplets);
    //int batch_size = min(BATCH_SIZE, total_triplets);
    int batch_size = BATCH_SIZE;
    int* d_triplet_deltas;
    int* d_node_triplets;

    hipMalloc(&d_triplet_deltas, batch_size * sizeof(int));
    hipMalloc(&d_node_triplets, batch_size * 4 * sizeof(int));

    int* h_triplet_deltas = (int*)malloc(batch_size * sizeof(int));
    int* h_node_triplets = (int*)malloc(batch_size * 4 * sizeof(int));



    // Initialize other variables
    int current_score = calculate_alignment_score(gm, gf, current_mapping);
    int best_score = current_score;
    int improvements = 0;
    time_t start_time = time(NULL);
    time_t last_sync_time = start_time;
    int last = 0, pass = 0;
    int iter=0;
    
    // Apply initial random perturbation
    //random_swap_k_vertices(current_mapping, max_node, 10 + rank, rank);
    //current_score = calculate_alignment_score(gm, gf, current_mapping);
    

    LOG_INFO("before loop, current score=%d, best score =%d", current_score, best_score);
    // Main optimization loop
    while (true) {
        // Synchronization block
        time_t current_time = time(NULL);
        

        if (difftime(current_time, last_sync_time) >= SYNC_INTERVAL) {
              LOG_INFO("Process %d, enter synchronization the best score is %d", rank,best_score);
              int* benchmark_mapping = load_benchmark_mapping("../data/best.csv", max_node);
              if (!benchmark_mapping) {
                  LOG_ERROR("Failed to load benchmark mapping");
              }

              int benchmark_score = calculate_alignment_score(gm, gf, benchmark_mapping);
              if (benchmark_score >best_score) {
                      best_score=benchmark_score;
                      memcpy(best_mapping, benchmark_mapping, sizeof(int) * (max_node + 1));
              }
              free(benchmark_mapping);


              last_sync_time = current_time;
              LOG_INFO("Process %d, after synchronization and best score is %d", rank,best_score);

        }



        // Copy current mapping to GPU
        hipMemcpy(d_mapping, current_mapping, (max_node + 1) * sizeof(int), hipMemcpyHostToDevice);
        
        bool found_improvement = false;
        long int batch_start = 0;
    
        while (batch_start < total_triplets) {
	    long int restsize=total_triplets - batch_start;
            int current_batch_size = batch_size;
	    if (restsize <batch_size ) {
		    current_batch_size =(int) restsize;
	    }
        
            // Launch kernel for current batch
            int threadsPerBlock = 256;
            int numBlocks = (current_batch_size + threadsPerBlock - 1) / threadsPerBlock;
        
            calculateThreeNodeSwapKernel<<<numBlocks, threadsPerBlock>>>(
                d_adj_matrix_m,
                d_adj_matrix_f,
                d_mapping,
                max_node,
                batch_start,
                current_batch_size,
                d_triplet_deltas,
                d_node_triplets
            );

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        LOG_ERROR("CUDA synchronize err: %s", hipGetErrorString(err));
        return NULL;
    }


            // Copy results back
            err=hipMemcpy(h_triplet_deltas, d_triplet_deltas, 
                   current_batch_size * sizeof(int), hipMemcpyDeviceToHost);


    if (err != hipSuccess) {
        LOG_ERROR("CUDA not initialized: %s", hipGetErrorString(err));
        return NULL;
    }


            err=hipMemcpy(h_node_triplets, d_node_triplets, 
                   current_batch_size * 4 * sizeof(int), hipMemcpyDeviceToHost);
        
    if (err != hipSuccess) {
        LOG_ERROR("CUDA not initialized: %s", hipGetErrorString(err));
        return NULL;
    }
            // Find best improvement in batch
            int best_delta = 0;
            int best_idx = -1;
        
            for (int i = 0; i < current_batch_size; i++) {
                if (h_triplet_deltas[i] > best_delta) {
                    best_delta = h_triplet_deltas[i];
                    best_idx = i;
                }
            }
        
            printf("max swap index =%d, max delta= %d from batch start %d\n",best_idx,best_delta,batch_start);
            // Apply best improvement if found
            if (best_delta > 0) {
                int node1 = h_node_triplets[best_idx * 4];
                int node2 = h_node_triplets[best_idx * 4 + 1];
                int node3 = h_node_triplets[best_idx * 4 + 2];
                int perm = h_node_triplets[best_idx * 4 + 3];
            
                apply_permutation(current_mapping, node1, node2, node3, perm);
                current_score += best_delta;
            
                if (current_score > best_score) {
                    memcpy(best_mapping, current_mapping, (max_node + 1) * sizeof(int));
                    best_score = current_score;
                    LOG_INFO("Process %d found new best score: %d", rank, best_score);
                    save_intermediate_mapping(out_path, best_mapping, max_node, gm, gf, best_score);
                }
            
                found_improvement = true;
		improvements++;
                break;  // Start over with new mapping
            }
        
            batch_start += (long int) current_batch_size + total_triplets/3;

        }
    
        // If no improvements found in any batch, do random perturbation
        if (!found_improvement) {
            memcpy(current_mapping, best_mapping, (max_node + 1) * sizeof(int));
            int num_pairs = 3 + rand() % 3;
            random_swap_k_vertices(current_mapping, max_node, num_pairs, rank);
            current_score = calculate_alignment_score(gm, gf, current_mapping);

            LOG_INFO("Process %d restarting optimization with perturbation pairs =%d, best score =%d", rank,num_pairs,best_score);
            LOG_INFO("after shuffle current_score=  %d", current_score);
	    improvements=0;
        }
        current_time = time(NULL);
	//if (iter++ % 2==0) {
	{   iter++;

            LOG_INFO("Average time for this loop is  %f for iteration %d", (current_time-start_time)/(iter*1.0),iter);
            LOG_INFO("Current score is %d  best score is %d of rank %d, improvement =%d", current_score, best_score, rank,improvements);
	}
    }
    
    // Cleanup
    hipFree(d_adj_matrix_m);
    hipFree(d_adj_matrix_f);
    hipFree(d_mapping);
    hipFree(d_triplet_deltas);
    hipFree(d_node_triplets);
    


    free(h_triplet_deltas);
    free(h_node_triplets);
    free(h_adj_matrix_m);
    free(h_adj_matrix_f);
    free(current_mapping);
    free(old_mapping);
    
    return best_mapping;
}

// Function to get maximum node ID from graph
int get_max_node(Graph* g) {
    return NUM_NODES;
}

// Function to clean up graph memory
void free_graph(Graph* g) {
    if (g->edges) {
        for (int i = 0; i < MAX_NODES; i++) {
            if (g->edges[i].count > 0) {
                free(g->edges[i].to_nodes);
                free(g->edges[i].weights);
            }
        }
        free(g->edges);
    }
    
    if (g->reverse_edges) {
        for (int i = 0; i < MAX_NODES; i++) {
            if (g->reverse_edges[i].count > 0) {
                free(g->reverse_edges[i].to_nodes);
                free(g->reverse_edges[i].weights);
            }
        }
        free(g->reverse_edges);
    }
    if (g->adj_matrix){
        for (int i=0; i<=NUM_NODES; i++){
            free(g->adj_matrix[i]);
        }
        free(g->adj_matrix);
        g->adj_matrix=NULL;
    }
    //free(g->nodes);
    free(g);
}

Graph* load_graph_from_csv(const char* filename) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        LOG_ERROR("Failed to open file: %s", filename);
        return NULL;
    }
    
    Graph* graph = new_graph();
    char line[MAX_LINE_LENGTH];
    int line_count = 0;
    int total_lines = 0;
    
    // Count total lines for progress bar
    while (fgets(line, MAX_LINE_LENGTH, file)) total_lines++;
    rewind(file);
    
    // Skip header
    fgets(line, MAX_LINE_LENGTH, file);
    total_lines--; // Adjust for header
    
    LOG_INFO("Loading graph from %s (%s lines)", filename, format_number(total_lines));
    
    time_t start_time = time(NULL);
    while (fgets(line, MAX_LINE_LENGTH, file)) {
        int from, to, weight;
        if (sscanf(line, "%d,%d,%d", &from, &to, &weight) == 3) {
            add_edge(graph, from, to, weight);
            line_count++;
            if (line_count % 100000 == 0) {
                print_progress(line_count, total_lines, "Loading graph");
            }
        } else {
            LOG_ERROR("Malformed line in CSV: %s", line);
        }
    }
    
    time_t end_time = time(NULL);
    LOG_INFO("Graph loaded successfully:");
    LOG_INFO("  - Nodes: %s", format_number(NUM_NODES));
    LOG_INFO("  - Edges: %s", format_number(line_count));
    LOG_INFO("  - Time taken: %ld seconds", end_time - start_time);
    
    fclose(file);
    return graph;
}

// Main function
int main(int argc, char* argv[]) {



    // Initialize CUDA first
    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess) {
        printf("hipSetDevice failed! Error: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Print device info
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Found %d CUDA devices\n", deviceCount);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using device: %s\n", prop.name);











    int rank=0;
    int size=4;

    if (argc < 5) {
        if (rank == 0) {
            LOG_ERROR("Usage: %s <male graph> <female graph> <in mapping> <out mapping>", argv[0]);
        }
        return 1;
    }
	

    time_t total_start = time(NULL);
    // Only rank 0 prints initial information
    if (rank == 0) {
        LOG_INFO("Graph Alignment Tool v1.0 (MPI + OpenMP)");
        LOG_INFO("Number of MPI processes: %d", size);
        LOG_INFO("Starting process with:");
        LOG_INFO("  - Male graph: %s", argv[1]);
        LOG_INFO("  - Female graph: %s", argv[2]);
        LOG_INFO("  - Output mapping: %s", argv[4]);
    }

    
    Graph* gm = load_graph_from_csv(argv[1]);
    if (!gm) {
        LOG_ERROR("Failed to load male graph");
        return 1;
    }
    
    Graph* gf = load_graph_from_csv(argv[2]);
    if (!gf) {
        LOG_ERROR("Failed to load female graph");
        free_graph(gm);
        return 1;
    }
    
    int max_node = MAX(get_max_node(gm), get_max_node(gf));
    
    int* benchmark = load_benchmark_mapping(argv[3], max_node);
    if (!benchmark) {
        LOG_ERROR("Failed to load benchmark mapping");
        free_graph(gm);
        free_graph(gf);
        return 1;
    }
    
    int initial_score = calculate_alignment_score(gm, gf, benchmark);
    LOG_INFO("Initial alignment score: %s", format_number(initial_score));
    
    char outputfilename[100];

    sprintf(outputfilename, "ID%d-%s", rank,argv[4]);

    sprintf(outputfilename, "%.*s-rank%d.csv", (int)strlen(argv[4])-4, argv[4], rank);
    LOG_INFO("output file name is %s",outputfilename);

    int* optimized_mapping = optimize_mapping(gm, gf, benchmark, outputfilename);
    int optimized_score = calculate_alignment_score(gm, gf, optimized_mapping);
    
    time_t total_end = time(NULL);
    LOG_INFO("Process completed:");
    LOG_INFO("  - Initial score: %s", format_number(initial_score));
    LOG_INFO("  - Final score: %s", format_number(optimized_score));
    LOG_INFO("  - Improvement: %.2f%%",
            (double)(optimized_score - initial_score) / initial_score * 100.0);
    LOG_INFO("  - Total time: %.1f minutes", difftime(total_end, total_start) / 60);
    
    free_graph(gm);
    free_graph(gf);
    free(benchmark);
    free(optimized_mapping);
    
    return 0;
}
